#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;


__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<n) C[i] = A[i] + B[i];
}

void print_vec(float* vector, int size)
{
    for(int i=0; i<size; i++)
        cout << vector[i] << " ";
    cout<<endl;
}

void vecAdd(float* A, float* B, float* C, int n)
{
    int size = n * sizeof(float);
    A = (float*)malloc(size);
    C = (float*)malloc(size);
    B = (float*)malloc(size);
    for( int i = 0; i < n; i++ ) {
        A[i] = sin(i)*cos(i);
        B[i] = sin(i)*sin(i);
    }

    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_C, size);
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
    vecAddKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    // Free device memory for A, B, C
    hipFree(d_A); hipFree(d_B); hipFree (d_C);
    print_vec(C, size);
}

int main()
{
    // Size of vectors
    int n = 50;
 
    // Host input vectors
    float* h_a;
    float* h_b;
    float* h_c;

    vecAdd(h_a, h_b, h_c, n);
    return 0;
}