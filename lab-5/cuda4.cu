#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

_global_ 
void thirdDimentionalKernel(float* d_in, float* d_out, int n, int m, int p) 
{
       int row = blockIdx.y*blockDim.y + threadIdx.y;
       int col = blockIdx.x*blockDim.x + threadIdx.x;
       int plane = blockIdx.z*blockDim.z + threadIdx.z;
       
       if ((plane < p)  && (col < n) && (row < m)) 
               d_Pout[plane * m * n + row * n + Col] = 2 * d_in[plane * m * n +row * n + Col];
}

void thirdDimentionalOperation(float* in ,float* out, int n, int m, int p)
{
       int size = p * m * n * sizeof(float);
       float* d_in, * d_out;

       hipMalloc((void **) &d_in, size);
       hipMalloc((void **) &d_out, size);
       
       hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
       
       thirdDimentionalKernel<<<(ceil(n/16), ceil(m/16), ceil(p/16)),(16, 16, 16)>>>(d_in, d_out, n, m, p);
               
       hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
       
       hipFree(d_in); 
       hipFree(d_out);
}

void initMat(float* mat, int n) {
    int i, j;
    for(i=0; i<n; i++)
        for(j=0; j<n; j++)
            mat[i*n+j] = rand() % 9 + 1;
}

void printMat(float *a, int n) {
    int i, j;
    for(int i=0; i<n; i++){
        for (int j=0; j<n; j++)
            cout<< a[i*n+j] << " ";
    cout<<endl;
    }    
}

int main(int argc, char const *argv[])
{
      int n = 100;
      float* in;
      float* out;

      initMat(in, n);
      initMat(out, n);

      thirdDimentionalOperation();
      printMat(out, n);
      return 0;
}