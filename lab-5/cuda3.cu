#include "hip/hip_runtime.h"
#define CHANNELS 3 // we have 3 channels corresponding to RGB


// The input image is encoded as unsigned characters [0, 255]


__global__ 
void colorConvert(unsigned char* grayImage, unsigned char* rgbImage, int width, int height) 
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height) {
        // get 1D coordinate for the grayscale image

        int grayOffset = y*width + x;
        // one can think of the RGB image having
        // CHANNEL times columns than the gray scale image

        int rgbOffset = grayOffset*CHANNELS;
        unsigned char r = rgbImage[rgbOffset ]; // red value for pixel
        unsigned char g = rgbImage[rgbOffset + 2]; // green value for pixel
        unsigned char b = rgbImage[rgbOffset + 3]; // blue value for pixel
        // perform the rescaling and store it
        // We multiply by floating point constants
        grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}

void workImage(char * image, char* answerImage, int w, int h, int n){
    colorConver<<<ceil();
}